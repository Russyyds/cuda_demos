//功能 
// stream1执行kernel1
// stream2等待kernel1执行完成并开始执行kernel2
// stream1执行完kernel1后继续执行kernel3
// stream1: kernel1....kernel3 
// stream2:           kernel2
//
#include <iostream>
#include <hip/hip_runtime.h>

// Kernel functions to perform computation
__global__ void kernel1(int64_t *data, int64_t repeat) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = 0; i < repeat; i++)
    {
        data[idx] += 1;
    }
}

__global__ void kernel2(int64_t *data, int64_t repeat) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = 0; i < repeat; i++)
    {
        data[idx] += 2;
    }
}

__global__ void kernel3(int64_t *data, int64_t repeat) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (size_t i = 0; i < repeat; i++)
    {
        data[idx] -= 1;
    }
}

int main() {
    const int data_size = 2048;
    const int print_size = 20;
    int64_t *host_data = new int64_t[data_size];
    int64_t *device_data1, *device_data2;
    for (int i = 0; i < data_size; i++) {
        host_data[i] = i;
    }

    hipMalloc((void**)&device_data1, data_size * sizeof(int64_t));
    hipMalloc((void**)&device_data2, data_size * sizeof(int64_t));

    hipMemcpy(host_data, device_data1, data_size * sizeof(int64_t), hipMemcpyHostToDevice);
    hipMemcpy(host_data, device_data2, data_size * sizeof(int64_t), hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((data_size + blockDim.x - 1) / blockDim.x);

    hipStream_t stream1, stream2;
    hipEvent_t event1;
    int high_priority, low_priority;
    hipDeviceGetStreamPriorityRange(&low_priority, &high_priority);
    hipStreamCreate(&stream1);
    hipStreamCreateWithPriority(&stream2, hipStreamDefault, high_priority);

    hipEventCreate(&event1);
    const int repeat = 1000;
    // 在stream1执行kernel1
    kernel1<<<gridDim, blockDim, 0, stream2>>>(device_data1, repeat);
    hipEventRecord(event1, stream1);
    hipStreamWaitEvent(stream2, event1);
    // 在stream2执行kernel2
    kernel2<<<gridDim, blockDim, 0, stream2>>>(device_data1, repeat);
    
    kernel3<<<gridDim, blockDim, 0, stream1>>>(device_data2, repeat);

    //同步两个stream
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipMemcpy(host_data, device_data1, data_size * sizeof(int64_t), hipMemcpyDeviceToHost);

    std::cout << "Data after kernel1 and kernel2:" << std::endl;
    for (int i = 0; i < print_size; i++) {
        std::cout << host_data[i] << " ";
    }
    std::cout << std::endl;

    hipMemcpy(host_data, device_data2, data_size * sizeof(int64_t), hipMemcpyDeviceToHost);

    std::cout << "Data after kernel3:" << std::endl;
    for (int i = 0; i < print_size; i++) {
        std::cout << host_data[i] << " ";
    }
    std::cout << std::endl;
    hipFree(device_data1);
    hipFree(device_data2);
    delete [] host_data;
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(event1);
    return 0;
}